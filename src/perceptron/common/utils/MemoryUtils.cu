#include "perceptron/common/utils/MemoryUtils.h"

namespace perceptron {
namespace utils {

bool
is_unregistered_host(const hipPointerAttribute_t &attrs) {
  return attrs.type == cudaMemoryTypeUnregistered;
}

bool
is_device(const hipPointerAttribute_t &attrs) {
  return attrs.type == hipMemoryTypeDevice;
}

bool
is_host(const hipPointerAttribute_t &attrs) {
  return attrs.type == hipMemoryTypeHost;
}

bool
is_managed(const hipPointerAttribute_t &attrs) {
  return attrs.type == hipMemoryTypeManaged;
}

} // perceptron
} // utils
