#include "perceptron/common/utils/CuBLASUtils.h"

namespace perceptron {
namespace utils {

hipblasOperation_t
trans2operation(bool trans) {
  return trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
}

hipblasOperation_t
inverse_trans(hipblasOperation_t trans) {
  return trans == HIPBLAS_OP_T ? HIPBLAS_OP_N : HIPBLAS_OP_T;
}

std::unique_ptr<hipblasHandle_t, CuBLASHandle::cublas_handle_deleter> CuBLASHandle::m_handle{};

hipblasHandle_t
CuBLASHandle::getInstance() {
  if (nullptr == m_handle) {
    m_handle = cublas_init_handle();
  }

  return *m_handle;
}

std::unique_ptr<hipblasHandle_t, CuBLASHandle::cublas_handle_deleter>
CuBLASHandle::cublas_init_handle() {
  std::unique_ptr<hipblasHandle_t> handle_ptr{new hipblasHandle_t};
  CUBLAS_CHECK(hipblasCreate(handle_ptr.get()));

  return std::unique_ptr<hipblasHandle_t, cublas_handle_deleter>{handle_ptr.release(), cublas_handle_deleter{}};
}

void CuBLASHandle::set_stream(hipStream_t stream) {
  CUBLAS_CHECK(hipblasSetStream(getInstance(), stream));
}

} // perceptron
} // utils
