#include "perceptron/common/utils/CuBLASUtils.h"

namespace perceptron {
namespace utils {

std::unique_ptr<hipblasHandle_t, CuBLASHandle::cublas_handle_deleter> CuBLASHandle::m_handle{};

hipblasHandle_t
CuBLASHandle::getInstance() {
  if (nullptr == m_handle) {
    m_handle = cublas_init_handle();
  }

  return *m_handle;
}

std::unique_ptr<hipblasHandle_t, CuBLASHandle::cublas_handle_deleter>
CuBLASHandle::cublas_init_handle() {
  std::unique_ptr<hipblasHandle_t> handle_ptr{new hipblasHandle_t};
  CUBLAS_CHECK(hipblasCreate(handle_ptr.get()));

  return std::unique_ptr<hipblasHandle_t, cublas_handle_deleter>{handle_ptr.release(), cublas_handle_deleter{}};
}

void CuBLASHandle::set_stream(hipStream_t stream) {
  CUBLAS_CHECK(hipblasSetStream(getInstance(), stream));
}

} // perceptron
} // utils
