#include "hip/hip_runtime.h"
#include "perceptron/common/utils/CurandUtils.h"

namespace perceptron {
namespace utils {
namespace details {

__global__
static void
curand_states_init(size_type seed, size_type size, hiprandState *state) {
  auto id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < size) {
    hiprand_init(seed, id, 0, &state[id]);
  }
}

} // details

const char *
curandGetErrorString(hiprandStatus_t error) {
  switch (error) {
  case HIPRAND_STATUS_SUCCESS:return "HIPRAND_STATUS_SUCCESS";
  case HIPRAND_STATUS_VERSION_MISMATCH:return "HIPRAND_STATUS_VERSION_MISMATCH";
  case HIPRAND_STATUS_NOT_INITIALIZED:return "HIPRAND_STATUS_NOT_INITIALIZED";
  case HIPRAND_STATUS_ALLOCATION_FAILED:return "HIPRAND_STATUS_ALLOCATION_FAILED";
  case HIPRAND_STATUS_TYPE_ERROR:return "HIPRAND_STATUS_TYPE_ERROR";
  case HIPRAND_STATUS_OUT_OF_RANGE:return "HIPRAND_STATUS_OUT_OF_RANGE";
  case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
  case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
  case HIPRAND_STATUS_LAUNCH_FAILURE:return "HIPRAND_STATUS_LAUNCH_FAILURE";
  case HIPRAND_STATUS_PREEXISTING_FAILURE:return "HIPRAND_STATUS_PREEXISTING_FAILURE";
  case HIPRAND_STATUS_INITIALIZATION_FAILED:return "HIPRAND_STATUS_INITIALIZATION_FAILED";
  case HIPRAND_STATUS_ARCH_MISMATCH:return "HIPRAND_STATUS_ARCH_MISMATCH";
  case HIPRAND_STATUS_INTERNAL_ERROR:return "HIPRAND_STATUS_INTERNAL_ERROR";
  }

  return "<unknown>";
}

CurandGeneratorOwner
curand_create_generator(size_type seed, hiprandRngType_t rng_type) {
  std::unique_ptr<hiprandGenerator_t> gen_ptr{new hiprandGenerator_t};
  CURAND_CHECK(hiprandCreateGenerator(gen_ptr.get(), rng_type));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(*gen_ptr, seed));
  return CurandGeneratorOwner{gen_ptr.release(), curand_generator_deleter_t{}};
}

CurandStatesOwner
curand_create_states(size_type seed, size_type size, hipStream_t stream) {
  auto states_ptr = cu_make_memory_unique<hiprandState_t>(size, stream);
  dim3 threads(utils::DEFAULT_BLOCK_SIZE_2D * utils::DEFAULT_BLOCK_SIZE_2D);
  dim3 blocks(utils::block_size_by_threads(size, threads.x));
  details::curand_states_init<<<blocks, threads, 0, stream>>>(seed, size, states_ptr.get());
  return states_ptr;
}

void
curand_set_stream(const CurandGeneratorOwner &gen,
                  hipStream_t stream) {
  CURAND_CHECK(hiprandSetStream(*gen, stream));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_uniform_tag tag,
                size_type n,
                float *ptr) {
  CURAND_CHECK(hiprandGenerateUniform(*gen, ptr, n));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_log_normal_tag tag,
                size_type n,
                float *ptr) {
  CURAND_CHECK(hiprandGenerateLogNormal(*gen, ptr, n, tag.mean, tag.stddev));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_normal_tag tag,
                size_type n,
                float *ptr) {
  CURAND_CHECK(hiprandGenerateNormal(*gen, ptr, n, tag.mean, tag.stddev));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_uniform_tag tag,
                size_type n,
                double *ptr) {
  CURAND_CHECK(hiprandGenerateUniformDouble(*gen, ptr, n));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_log_normal_tag tag,
                size_type n,
                double *ptr) {
  CURAND_CHECK(hiprandGenerateLogNormalDouble(*gen, ptr, n, tag.mean, tag.stddev));
}

void
curand_generate(const CurandGeneratorOwner &gen,
                curand_normal_tag tag,
                size_type n,
                double *ptr) {
  CURAND_CHECK(hiprandGenerateNormalDouble(*gen, ptr, n, tag.mean, tag.stddev));
}

} // perceptron
} // utils
